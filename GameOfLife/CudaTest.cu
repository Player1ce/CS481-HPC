#include <stdio.h>
#include <iostream>

__global__ void helloCUDA() {
    printf("Hello, CUDA!\n");
}


#include <hip/hip_runtime.h>
#include <iostream>

__device__ int update_flag = 0;

// Kernel to set the flag using atomic operations
__global__ void setFlag() {
    atomicExch(&update_flag, 1);
}
// Kernel to set the flag using atomic operations
__global__ void resetFlag() {
    atomicExch(&update_flag, 0);
}

// Kernel to check and print the value of the flag
__global__ void checkFlag() {
    printf("Flag value in device: %d\n", update_flag);
}

int main() {
    helloCUDA<<<1, 1>>>(); // Launch kernel with 1 block and 10 threads
    hipDeviceSynchronize(); // Wait for the GPU to finish

    int deviceId = 0; // Assuming you want to query the first GPU
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);

    std::cout << "Device Name: " << deviceProp.name << std::endl;
    std::cout << "Device ID: " << deviceId << std::endl;

    std::cout << "Number of SMs: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Max blocks per SM: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "Max threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;


    // Allocate memory for the device flag
    int* d_update_flag;
    hipMalloc(&d_update_flag, sizeof(int));

    // Initialize the flag to 0 using cudaMemset
    hipMemset(d_update_flag, 0, sizeof(int));
    std::cout << "Initial flag value set to 0 via hipMemset" << std::endl;

    // Launch kernel to set the flag
    setFlag<<<1, 1>>>();
    hipDeviceSynchronize();

    // Launch kernel to check the flag
    checkFlag<<<1, 1>>>();
    hipDeviceSynchronize();

    // Reset the flag using cudaMemset
    hipMemset(d_update_flag, 0, sizeof(int));
    std::cout << "Flag reset to 0 via hipMemset" << std::endl;
    resetFlag<<<1,1>>>();

    // Launch kernel to check the flag again
    checkFlag<<<1, 1>>>();
    hipDeviceSynchronize();

    // Free the allocated memory
    hipFree(d_update_flag);

    return 0;
}
