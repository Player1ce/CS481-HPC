
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void helloCUDA() {
    printf("Hello, CUDA!\n");
}

int main() {
    helloCUDA<<<1, 1>>>(); // Launch kernel with 1 block and 10 threads
    hipDeviceSynchronize(); // Wait for the GPU to finish

    int deviceId = 0; // Assuming you want to query the first GPU
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);

    std::cout << "Device Name: " << deviceProp.name << std::endl;
    std::cout << "Device ID: " << deviceId << std::endl;

    std::cout << "Number of SMs: " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "Max blocks per SM: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "Max threads per SM: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;

    return 0;
}
