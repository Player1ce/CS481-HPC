#include "hip/hip_runtime.h"
//
// Created by motst on 10/3/2024.
//

#include "LibraryCode.hpp"
#include "FileIO.hpp"

#ifdef _OPENMP
# include <omp.h>
#endif

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <random>
#include <iostream>
#include <sstream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <atomic>
#include <thread>

using namespace std;
using namespace util;

#define CELL_UPDATE_DEBUG_LOGGING

template<typename T>
void fillFromVector(T **array, const int rows, const int columns, const vector<bool>& list, const int border) {
    for (int i = border; i < rows + border; i++) {
        for (int j = border; j < columns + border; j++)
            array[i][j] = list.at(columns * (i - border) + (j - border));
    }
}

template<typename T>
void fillWithRandom(T **array, const int rows, const int columns, const int border, const int min = 0,
                    const int max = 1, const bool useRandom = true) {
    // Create a random number generator

    if (useRandom) {
        std::random_device seed;
        std::mt19937 generator(seed());

        // Create a distribution for your desired range
        std::uniform_int_distribution<int> distribution(min, max);

        for (int i = border; i < rows + border; i++) {
            for (int j = border; j < columns + border; j++)
                array[i][j] = distribution(generator);
        }
    } else {
        std::mt19937 generator(12345);

        // Create a distribution for your desired range
        std::uniform_int_distribution<int> distribution(min, max);

        for (int i = border; i < rows + border; i++) {
            for (int j = border; j < columns + border; j++)
                array[i][j] = distribution(generator);
        }
    }
}

template<typename T>
int getSum(T **array, const int rows, const int columns, const int border) {
    int sum = 0;
    for (int i = border; i < rows + border; i++) {
        for (int j = border; j < columns + border; j++) {
            sum += array[i][j];
        }
    }
    return sum;
}

template<typename T>
std::string arrayToString(T **array, const int rows, const int columns, const int border) {
    std::stringstream msg("");
    msg << "[\n";
    for (int i = border; i < rows + border; i++) {
        msg << "[ ";
        for (int j = border; j < columns + border; j++) {
            msg << array[i][j] << " ";
        }
        msg << "]\n";
    }
    msg << "]\n";
    return msg.str();
}

void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


vector<bool> initializer = {
    true, true, false, false, true,
    false, true, true, false, false,
    true, true, false, false, true,
    false, true, true, true, true,
    true, true, true, false, true
};

vector<bool> test = {
    true, true, true, false, false,
    false, false, true, true, false,
    true, false, false, false, true,
    false, false, false, false, true,
    true, false, false, false, true,
};

vector<bool> test_secondIteration = {
    false, true, true, true, false,
    true, false, true, true, false,
    false, false, false, false, true,
    false, false, false, true, true,
    false, false, false, false, false,
};


vector<bool> initializer2 = {
    false, false, false, false, false,
    false, false, false, false, false,
    false, false, true, false, false,
    false, false, false, false, false,
    false, false, false, false, false
};

auto tester2 = {
    false, false, false, false, false,
    false, false, false, false, false,
    false, false, false, false, false,
    false, false, false, false, false,
    false, false, false, false, false
};

// TODO:  store cols no updates in shared memory
// TODO:  coalesce cols no updates efficiently using tree architecture?
// TODO:  store offset calculation in shared memory and only update in master thread
__global__ void standard_check_cuda(int *boards, const int board_rows, const int board_columns, int board_border,
                                    int offset, int nextOffset) {

    int allocation_rows = board_rows + 2 * board_border;
    int allocation_columns = board_columns + 2 * board_border;

    // printf("r: %d, c: %d : %d %c", (blockIdx.y * blockDim.y) + threadIdx.y , (blockIdx.x * blockDim.x) + threadIdx.x, boards[(threadIdx.y + board_border) * allocation_columns + threadIdx.x + 1],
    // '\n');

    // printf("blockx: %d, blocky: %d \n", blockIdx.x, blockIdx.y);
    int InnerCellsNoUpdate = 0;


    int calculatedRow = (blockIdx.y * blockDim.y) + threadIdx.y;
    int calculatedColumn = (blockIdx.x * blockDim.x) + threadIdx.x;


    int rawIndex = (calculatedRow + board_border) * allocation_columns +
                   /* account for vertical displacement */
                   board_border + /* border offset for the row won't get accounted for in vertical displacement */
                   calculatedColumn; /* account for thread horizontal displacement */


    int index = (offset * (allocation_rows * allocation_columns)) + /* account for offset */
                rawIndex;

    // printf("rawIndex: %d | index = %d | index limit: %d \n",
    //         rawIndex, index,
    //         allocation_rows * allocation_columns - (board_border + allocation_columns));


    // __syncthreads();

    int update = 0;
    if (calculatedRow < board_rows && calculatedColumn < board_columns) {
        // TODO: test this
        // int lowerRow = index + allocation_columns;
        // int upperRow = index - allocation_columns;

        int value = boards[index - allocation_columns - 1] + boards[index - allocation_columns] + boards[index - allocation_columns + 1]
                    + boards[index - 1] + boards[index + 1]
                    + boards[index + allocation_columns  - 1] + boards[index + allocation_columns] + boards[index + allocation_columns + 1];

        int oldVal = boards[index];
        int newVal = (value == 3) ? 1 : (value == 2) ? oldVal : 0;

        // set next cell
        boards[(nextOffset * (allocation_rows * allocation_columns)) + rawIndex] = newVal;

        // TODO: implement efficient update collection. Otherwise this will serialize
        update = oldVal == newVal;
    }


    // TODO: implement check for master thread
    // innerRowsNoUpdates += innerColsNoUpdates == columns;
    // innerColsNoUpdates = 0;
    //                cout << endl;
}

// theirs
//5000x1000: 46.3

// standalone checked
// 5000x1000: 25.7, 24.6


// standalone 10 thread checked
// 5000x1000: 11.4
// 5000x5000: 54, 57.185
// 10000x10000: 451.659


//#define EARLY_STOP_LOGGING


// #define STANDARD_CHECK
// #define STANDARD_CHECK_OMP
#define STANDARD_CHECK_CUDA
//#define STANDARD_CHECK_CUDA_NONBLOCKING

int main(int argc, char **argv) {
    //    initializer = initializer2;
    //    test = tester2;

    int rows = 1000;
    int columns = rows;

    int iterations = 1000;

    bool useInitializerList = false;

    int offset = 0;
    int nextOffset = 1;
    constexpr int maxOffset = 1;

    constexpr int printThreshold = 50;

    constexpr int border = 1;
    constexpr int numArrays = 2;

    constexpr bool useRandom = false;

    bool writeToFile = false;
    std::string outputDirectory;

    bool useTestFile = false;
    std::string testFile;

    int numThreads = 5;

    // region CLI_arguments

    if (argc < 2) {
        cout << "Using coded constants" << endl;
    } else if (argc == 2) {
        cout << "Using Vector initializer" << endl;
        useInitializerList = true;
        rows = columns = 5;
        iterations = 1;
        numThreads = 1;
    } else if (argc == 3) {
        cout << "Using size: " << argv[1] << " and iterations: " << argv[2] << endl;
        rows = atoi(argv[1]);
        columns = rows;
        iterations = atoi(argv[2]);
    } else if (argc == 4) {
        cout << "Using size: " << argv[1] << " and iterations: " << argv[2] << " and numThreads: " << argv[3] << endl;
        rows = atoi(argv[1]);
        columns = rows;
        iterations = atoi(argv[2]);
        numThreads = atoi(argv[3]);
    } else if (argc == 5) {
        cout << "Using rows: " << argv[1] << " and iterations: " << argv[2] << " and numThreads: " << argv[3] <<
                " and filePath: " << argv[4] << std::endl;
        rows = atoi(argv[1]);
        columns = rows;
        iterations = atoi(argv[2]);
        numThreads = atoi(argv[3]);
        outputDirectory = argv[4];
        writeToFile = true;
    } else if (argc == 6) {
        cout << "Using rows: " << argv[1] << " and iterations: " << argv[2] << " and numThreads: " << argv[3] <<
                " and filePath: " << argv[4] << " and test file name:" << argv[5] << std::endl;
        rows = atoi(argv[1]);
        columns = rows;
        iterations = atoi(argv[2]);
        numThreads = atoi(argv[3]);
        outputDirectory = argv[4];
        writeToFile = true;

        testFile = argv[5];
        useTestFile = true;
    }

    if (rows == columns && columns == 5) {
        useInitializerList = true;
    }
    // endregion CLI_arguments

    int printCount = max(iterations / 10, 1);

    int ***_arrays = new int **[2];
    int **_blocks = new int *[2];

    int allocation_rows = rows + (2 * border);
    int allocation_cols = columns + (2 * border);
    int allocation_size = allocation_rows * allocation_cols;

    cout << "allocation size: " << allocation_size << endl;

    // create the arrays and their borders. Also store the blocks for easy transfer to Cuda
    for (int i = 0; i < numArrays; i++) {
        int *block;
        int **grid;

        block = new int[allocation_rows * allocation_cols];
        grid = new int *[allocation_rows];

        for (int j = 0; j < allocation_rows; j++) {
            grid[j] = &block[j * allocation_cols];
        }

        _arrays[i] = grid;
        _blocks[i] = block;

        // Create the border
        for (int row = 0; row < rows + 2 * border; row++) {
            for (int colInset = 0; colInset < border; colInset++) {
                _arrays[i][row][colInset] = 0;
                _arrays[i][row][columns + border - colInset] = 0;
            }
        }

        for (int col = 0; col < columns + 2 * border; col++) {
            for (int rowInset = 0; rowInset < border; rowInset++) {
                _arrays[i][rowInset][col] = 0;
                _arrays[i][rows + border - rowInset][col] = 0;
            }
        }
    }

    // fill with random
    fillWithRandom(_arrays[0], rows, columns, border, 0, 1, useRandom);

    if (useInitializerList) {
        fillFromVector(_arrays[0], rows, columns, initializer, border);
    }

    const int sum = getSum(_arrays[0], rows, columns, border);

    if (rows * columns < printThreshold * printThreshold) {
        cout << arrayToString(_arrays[0], rows, columns, border) << endl;
    }

    int colsNoUpdates = 0;
    int rowsNoUpdates = 0;

    bool updateOccurred = true;

    chrono::time_point<chrono::system_clock> start, end;

    // region cuda_initialization
    int *d_boards;
    checkCudaError(hipMalloc(reinterpret_cast<void **>(&d_boards), sizeof(int) * allocation_size * 2));

    // get gpu properties
    int deviceId = 0; // Assuming you want to query the first GPU
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, deviceId);

    int min_block_size = 128;
    int min_width = 32;
    int min_height = 4;

    // create dimensions for Cuda code:
    dim3 gridDimensions_2D;
    dim3 blockDimensions_2D;

    if (rows * columns < min_block_size) {
        blockDimensions_2D.x = rows;
        blockDimensions_2D.y = columns;
    } else {
        blockDimensions_2D.x = min_width;
        blockDimensions_2D.y = min_height;
    }

    if (rows < blockDimensions_2D.y) {
        blockDimensions_2D.y = rows;
    }

    // blockDimensions_2D.x = 1;
    // blockDimensions_2D.y = 1;

    gridDimensions_2D.x = std::ceil(static_cast<float>(rows) / static_cast<float>(blockDimensions_2D.x));
    gridDimensions_2D.y = std::ceil(static_cast<float>(columns) / static_cast<float>(blockDimensions_2D.y));

    int minGridSize;
    int blockSize;

    // hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, standard_check_cuda, 0, 0);
    // cout << "minGridSize: " << minGridSize << endl;
    // cout << "blockSize: " << blockSize << endl;

    cout << "grid_x: " << gridDimensions_2D.x << endl;
    cout << "grid_y: " << gridDimensions_2D.y << endl;

    cout << "block_x: " << blockDimensions_2D.x << endl;
    cout << "block_y: " << blockDimensions_2D.y << endl;


    // endregion
    start = chrono::system_clock::now();


    // Copy first block from host to device
    checkCudaError(hipMemcpy(d_boards, _blocks[0], sizeof(int) * allocation_size, hipMemcpyHostToDevice));

    // Copy second block from host to device
    checkCudaError(hipMemcpy(d_boards + allocation_size, _blocks[1], sizeof(int) * allocation_size,
                              hipMemcpyHostToDevice));

    int currentIteration = 0;
    for (currentIteration = 0; currentIteration < iterations; currentIteration++) {
        // tested on 1000x1000 for 1000 iterations with 1 thread

        // standard_no_check 0.5
        // standard_check 0.8
        // standard_no_check_omp 0.55
        // standard_check_omp: 0.87

        // laptop
        // standard_nocheck: .935
        // standard_check_omp_test: 0.89


        // region standard_check
#ifdef STANDARD_CHECK
        updateOccurred = false;
        colsNoUpdates = 0;
        rowsNoUpdates = 0;

        // mine (full check): 0.8
        // mine (every time):
        // theirs: 1.8 with bool;  1.9 with flag
        //
        // method 1updateOccurred = (oldVal != newVal) || updateOccurred;: 2.5
        // method 2if (!updateOccurred && oldVal != newVal) updateOccurred = true;: 2.3
        // flag method: 2.15
        // flag += oldVal != newVal; : 2.2
        //

        for (int row = border; row < rows + border; row++) {
            for (int column = border; column < columns + border; column++) {
                int value = _arrays[offset][row - 1][column - 1] + _arrays[offset][row - 1][column] +
                            _arrays[offset][row - 1][column + 1]
                            + _arrays[offset][row][column - 1] + _arrays[offset][row][column + 1]
                            + _arrays[offset][row + 1][column - 1] + _arrays[offset][row + 1][column] +
                            _arrays[offset][row + 1][column + 1];

                int oldVal = _arrays[offset][row][column];
                int newVal = (value == 3) ? 1 : (value == 2) ? oldVal : 0;

                _arrays[nextOffset][row][column] = newVal;
                colsNoUpdates += oldVal == newVal;
            }
            rowsNoUpdates += colsNoUpdates == columns;
            colsNoUpdates = 0;
        }
#endif
        // endregion


        // region standard_check_omp
#ifdef STANDARD_CHECK_OMP
        rowsNoUpdates = 0;

        #pragma omp parallel num_threads(numThreads) \
            default(none) \
            shared(_arrays, rows, columns, offset, nextOffset, groups, rowsNoUpdates, cout)

        {
            int my_rank;
            int innerRowsNoUpdates = 0;
            int innerColsNoUpdates = 0;

#ifdef _OPENMP
            my_rank = omp_get_thread_num();
//            cout << "my rank: " << my_rank << endl;
#else
            my_rank = 0;
#endif

//            this_thread::sleep_for(chrono::milliseconds(my_rank * 100));

            for (int row = groups.at(my_rank).first + border; row < groups.at(my_rank).second + border; row++) {
                for (int column = border; column < columns + border; column++) {

                    int value = _arrays[offset][row - 1][column - 1] + _arrays[offset][row - 1][column] +
                              _arrays[offset][row - 1][column + 1]
                              + _arrays[offset][row][column - 1] + _arrays[offset][row][column + 1]
                              + _arrays[offset][row + 1][column - 1] + _arrays[offset][row + 1][column] +
                              _arrays[offset][row + 1][column + 1];

                    int oldVal = _arrays[offset][row][column];
                    int newVal = (value == 3) ? 1 : (value == 2) ? oldVal : 0;


                    _arrays[nextOffset][row][column] = newVal;
                    innerColsNoUpdates += oldVal == newVal;

//                    cout << "[" << row << ", " << column << ", s:" << sum << "] ";
                }
                innerRowsNoUpdates += innerColsNoUpdates == columns;
                innerColsNoUpdates = 0;
//                cout << endl;
            }

            #pragma omp critical
            {
                rowsNoUpdates += innerRowsNoUpdates;
            }
        }

#endif
        // endregion

        //region standard_check_cuda
#ifdef STANDARD_CHECK_CUDA

        standard_check_cuda<<<gridDimensions_2D,blockDimensions_2D>>>(d_boards, rows, columns, border, offset, nextOffset);

        checkCudaError(hipDeviceSynchronize());

#endif
        // endregion

        offset = nextOffset;
        nextOffset = (offset + 1) % (maxOffset + 1);

        if (rowsNoUpdates == rows) {
            cout << "exiting early on iteration: " << currentIteration + 1 << " because there was no update" << endl;
            break;
        }

        //        if (i == printCount * multiplier) {
        //            cout << "On iteration: " << currentIteration << " , " << (i/static_cast<double>(iterations))*100.0  << "%" << endl;
        //              multiplier++;
        //        }
    }

    end = chrono::system_clock::now();

    long seconds = chrono::duration_cast<chrono::seconds>(end - start).count();
    long ms = chrono::duration_cast<chrono::milliseconds>(end - start).count();
    long decimal = ms - seconds * 1000;
    cout << "System took: " << seconds << "." << decimal << " seconds to run" << endl;

#ifdef STANDARD_CHECK
    // Copy current block from host to device
    checkCudaError(hipMemcpy(d_boards + (allocation_size * offset), _blocks[offset], sizeof(int) * allocation_size,
                              hipMemcpyHostToDevice));

#endif

    // test Memcpy. If using the test case this will cause the result to be wrong unless the board copies correctly.
    if (useInitializerList) {
        cout << "Editing element to make test fail if Memcpy doesn't work." << endl;
        _arrays[offset][3][3] = 1;
    }

    // Copy cuda data back to array
    checkCudaError(hipMemcpy(_blocks[offset], d_boards + (allocation_size * offset),
                              sizeof(int) * allocation_size, hipMemcpyDeviceToHost));


    if (rows * columns < printThreshold * printThreshold) {
        cout << "end matrix " << endl << arrayToString(_arrays[offset], rows, columns, border);
    }

    if (useInitializerList) {
        if (rows * columns < printThreshold * printThreshold) {
            if (iterations == 2) {
                test = test_secondIteration;
            }

            bool success = true;

            for (int i = 0; i < rows; i++) {
                for (int j = 0; j < columns; j++) {
                    cout << "[t:" << test.at((i * rows) + j) << ", r:" << (_arrays[offset][i + border][j + border] == 1
                                                                               ? 1
                                                                               : 0) << "] ";
                    if (test.at((i * rows) + j) != (_arrays[offset][i + border][j + border] == 1 ? 1 : 0)) {
                        success = false;
                    }
                }
                std::cout << std::endl;
            }

            cout << "Success: " << boolalpha << success << endl;
        }
    }

    const double percent = (sum / static_cast<double>(rows * columns)) * 100.0;
    cout << "percent: " << percent << "%" << endl;

    if (writeToFile) {
        if (outputDirectory.back() != '/') {
            outputDirectory.append("/");
        }

        std::stringstream fileName;
        fileName << "output_" << rows << "x" << columns << "_" << iterations << "_";

        int fileNum = 0;

        auto filesInDirectory = file_io::listDirectory(outputDirectory);

        //        cout << "Files in dir: ";
        //        for (const auto& file : filesInDirectory) {
        //            cout << file << " | ";
        //        }
        //        cout << endl;

        //        cout << "test: " << outputDirectory << fileName.str() + std::to_string(fileNum) + ".txt" << endl;

        while (std::find(filesInDirectory.begin(), filesInDirectory.end(),
                         outputDirectory + fileName.str() + std::to_string(fileNum) + ".txt") != filesInDirectory.
               end()) {
            fileNum++;
        }

        fileName << fileNum << ".txt";

        if (file_io::writeTofile(outputDirectory + fileName.str(),
                                 {arrayToString(_arrays[offset], rows, columns, border)})) {
            cout << "successfully wrote output to file: " << outputDirectory << fileName.str() << endl;
        } else {
            cout << "Failed to write to file: " << outputDirectory << fileName.str() << endl;
        }
    }

    if (useTestFile) {
        std::string fileContents = file_io::readFullFile(outputDirectory + testFile);
        //        cout << "file contents: " << endl << fileContents << endl;
        //        cout << "result of to string: " << endl << arrayToString(_arrays[offset], rows, columns, border) << endl;
        bool success = fileContents == arrayToString(_arrays[offset], rows, columns, border) + "\n";
        cout << boolalpha << "Test file was the same: " << success << endl;
    }

    for (int i = 0; i < maxOffset; i++) {
        LibraryCode::deleteArray(_arrays[i]);

        delete[] _arrays;
    }

    // free cuda memory
    checkCudaError(hipFree(d_boards));


    return 0;
}
